#ifndef _KERNELS
#define _KERNELS

//#include "types.h"

#define BLOCK_SIZE 32

__global__ void forwardKernel(const int m,const int n,const double *A, const double *x, double *y, double *deriv){
	int j;
	double Cval=0;
	//int row = blockIdx.y * blockDim.y + threadIdx.y;
  //int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row<m){
		for(j=0;j<n;j++){
			Cval+=A[row*n+j]*x[j];
		}
		double tmp=tanhf(Cval);
		y[row]=tmp;
		deriv[row]=1.0-(tmp*tmp);
	}
}
void forwardGPU(const int m,const int n,const double *A, const double *x, double *y, double *deriv){
    size_t size = m * n * sizeof(double);
		double *d_A=(double *)malloc(size);
    hipMalloc(&d_A,size);
    hipMemcpy(d_A,A,size,
               hipMemcpyHostToDevice);

		size_t xSz=n*sizeof(double);
		double *d_x=(double *)malloc(xSz);
    hipMalloc(&d_x,xSz);
    hipMemcpy(d_x,x,xSz,
	    hipMemcpyHostToDevice);
		
		size_t ySz=m*sizeof(double);
		double *d_y=(double *)malloc(ySz);
    hipMalloc(&d_y,ySz);
		
		double *d_deriv=(double *)malloc(ySz);
    hipMalloc(&d_deriv,ySz);

    // Invoke kernel
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(B.m / dimBlock.x, A.n / dimBlock.y);
    //dim3 dimGrid(dimBlock.x, A.n / dimBlock.y);

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((int)ceil((float)m/(float)dimBlock.x));
    //dim3 dimBlock(m);
    //dim3 dimGrid(1);
    forwardKernel<<<dimGrid, dimBlock>>>(m,n,d_A, d_x, d_y, d_deriv);

    // Read from device memory
    hipMemcpy(y,d_y,ySz,
    	hipMemcpyDeviceToHost);
    
		hipMemcpy(deriv,d_deriv,ySz,
    	hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_deriv);
}

#endif
