#ifndef _KERNELS
#define _KERNELS

//#include "types.h"

#define BLOCK_SIZE 32
#define GAMMA (0.1)

__global__ void forwardKernel(const int m,const int n,const double *A, const double *x, double *y, double *deriv){
	int j;
	double Cval=0;
	//int row = blockIdx.y * blockDim.y + threadIdx.y;
  //int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row<m){
		for(j=0;j<n;j++){
			Cval+=A[row*n+j]*x[j];
		}
		double tmp=tanhf(Cval);
		y[row]=tmp;
		deriv[row]=1.0-(tmp*tmp);
	}
}
void forwardGPU(const int m,const int n,const double *A, const double *x, double *y, double *deriv){
    size_t size = m * n * sizeof(double);
		double *d_A=(double *)malloc(size);
    hipMalloc(&d_A,size);
    hipMemcpy(d_A,A,size,
               hipMemcpyHostToDevice);

		size_t xSz=n*sizeof(double);
		double *d_x=(double *)malloc(xSz);
    hipMalloc(&d_x,xSz);
    hipMemcpy(d_x,x,xSz,
	    hipMemcpyHostToDevice);
		
		size_t ySz=m*sizeof(double);
		double *d_y=(double *)malloc(ySz);
    hipMalloc(&d_y,ySz);
		
		double *d_deriv=(double *)malloc(ySz);
    hipMalloc(&d_deriv,ySz);

    // Invoke kernel
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(B.m / dimBlock.x, A.n / dimBlock.y);
    //dim3 dimGrid(dimBlock.x, A.n / dimBlock.y);

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((int)ceil((float)m/(float)dimBlock.x));
    //dim3 dimBlock(m);
    //dim3 dimGrid(1);
    forwardKernel<<<dimGrid, dimBlock>>>(m,n,d_A, d_x, d_y, d_deriv);

    // Read from device memory
    hipMemcpy(y,d_y,ySz,
    	hipMemcpyDeviceToHost);
    
		hipMemcpy(deriv,d_deriv,ySz,
    	hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_deriv);
}

__global__ void deltaKernel(const int m,const int n,const double *A,double *delta,const double *y,const double *deriv){
	int i;
	double Cval=0;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(col<n){
		for(i=0;i<m;i++){
			Cval+=A[i*n+col]*y[i];
		}
		delta[col]=deriv[col]*Cval;
	}
}
void deltaGPU(const int m,const int n,const double *A,double *delta,const double *y,const double *deriv){
    size_t size = m * n * sizeof(double);
		double *d_A=(double *)malloc(size);
    hipMalloc(&d_A,size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);

		size_t xSz=n*sizeof(double);
		double *d_delta=(double *)malloc(xSz);
    hipMalloc(&d_delta,xSz);
//    cudaMemcpy(d_delta,delta,xSz,cudaMemcpyHostToDevice);
		
		size_t ySz=m*sizeof(double);
		double *d_y=(double *)malloc(ySz);
    hipMalloc(&d_y,ySz);
    hipMemcpy(d_y,y,ySz,hipMemcpyHostToDevice);
		
		double *d_deriv=(double *)malloc(xSz);
    hipMalloc(&d_deriv,xSz);
    hipMemcpy(d_deriv,deriv,xSz,hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((int)ceil((float)n/(float)dimBlock.x));
    //dim3 dimBlock(m);
    //dim3 dimGrid(1);
    deltaKernel<<<dimGrid,dimBlock>>>(m,n,d_A,d_delta,d_y,d_deriv);

    hipMemcpy(delta,d_delta,xSz,hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_delta);
    hipFree(d_y);
    hipFree(d_deriv);
}
__global__ void weightKernel(const int m,const int n,double *A,double *x,double *delta){
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	if((row<m)&&(col<n)){
		A[row*n+col] -= GAMMA*x[col]*delta[row];
	}
}
__global__ void batchAccumKernel(const int m,const int n,double *A,double *x,double *delta){
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	if((row<m)&&(col<n)){
		A[row*n+col] += GAMMA*x[col]*delta[row];
	}
}
__global__ void batchUpdateKernel(const int m,const int n,double *A,double *dA){
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	if((row<m)&&(col<n)){
		A[row*n+col] -= dA[row*n+col];
	}
}
__global__ void batchUpdateDMKernel(const int m,const int n,double *A,double *dA){
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	if((row<m)&&(col<n)){
		double x;
		x=dA[row*n+col];
		if(x>1)
			A[row*n+col] -= 1;
		else if(x<(-1))
			A[row*n+col] += 1;
		else
			A[row*n+col] -= x;
	}
}

#endif
