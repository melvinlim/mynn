#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define LAYERS 3
#define L1N 5
#define L1M 7
#define L2N 40
#define L2M 40
#define L3N 40
#define L3M 40

const int nDim[LAYERS]={L1N,L2N,L3N};
const int mDim[LAYERS]={L1M,L2M,L3M};

struct Layer{
	float **L;
	int *N;
	int *M;
};
struct Net{
	Layer **L;
};
void PRINTMATRIX(float **x,int n,int m){
	int i,j;
	for(i=0;i<n;i++){
		for(j=0;j<m;j++){
			printf("[%i,%i]%.02f\t",i,j,x[i][j]);
		}
	}
	printf("\n");
}
void PRINTARRAY(float *x,int sz){
	int i;
	for(i=0;i<sz;i++){
		printf("[%i]%.02f\t",i,*x++);
	}
	printf("\n");
}
int main(){
	int i,j,k;
	float test[L1N*L1M];
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	for(i=0;i<LAYERS;i++){
		net->L[i]=(Layer *)malloc(sizeof(Layer));
		net->L[i]->N=(int *)malloc(sizeof(int));
		net->L[i]->M=(int *)malloc(sizeof(int));
		memcpy(net->L[i]->N,&nDim[i],sizeof(int));
		memcpy(net->L[i]->M,&mDim[i],sizeof(int));
		net->L[i]->L=(float **)malloc(nDim[i]*sizeof(float *));
		for(j=0;j<nDim[i];j++){
			net->L[i]->L[j]=(float *)malloc(mDim[i]*sizeof(float));
		}
	}
	for(i=0;i<L1N*L1M;i++){
		test[i]=i;
	}
	PRINTARRAY(test,L1N*L1M);
	PRINTARRAY(test,L1N*L1M);
	PRINTMATRIX(net->L[0]->L,*net->L[0]->N,*net->L[0]->M);
	//memcpy(net->L[0]->L,test,L1N*L1M*sizeof(float));
	k=0;
	for(i=0;i<*net->L[0]->N;i++){
		for(j=0;j<*net->L[0]->M;j++){
			net->L[0]->L[i][j]=k++;
		}
	}
	PRINTMATRIX(net->L[0]->L,*net->L[0]->N,*net->L[0]->M);
}
