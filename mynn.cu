#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "matrixmul.cu"

#define LAYERS 3
#define L1N 6
#define L1M 6
#define L2N 6
#define L2M 40
#define L3N 40
#define L3M 40

#define RANDSCALING 10	//scale random weights to be from -0.1 to +0.1

const int nDim[LAYERS]={L1N,L2N,L3N};
const int mDim[LAYERS]={L1M,L2M,L3M};

struct Layer{
	Array *in;
	Matrix *M;
	Array *out;
};
struct Net{
	Layer **L;
	int size;
};
void PRINTMATRIX(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			printf("[%i,%i]%.04f\t",i,j,M->elements[i*M->stride+j]);
		}
	}
	printf("\n");
}
void PRINTARRAY(Array *A){
	int i;
	float *x;
	int sz=A->len;
	x=A->el;
	for(i=0;i<sz;i++){
		printf("[%i]%.02f\t",i,*x++);
	}
	printf("\n");
}
void randMatrix(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			M->elements[i*M->stride+j]=
			(random()-(RAND_MAX/2))*2.0/((float)RAND_MAX)/((float)RANDSCALING);
		}
	}
}
void randArray(Array *A){
	int i;
	for(i=0;i<A->len;i++){
		A->el[i]=
		(random()-(RAND_MAX/2))*2.0/((float)RAND_MAX)/((float)RANDSCALING);
	}
}
void nnRand(Net *N){
	Matrix *pM;
	int i;
	int n=N->size;
	for(i=0;i<n;i++){
		pM=N->L[i]->M;
		randMatrix(pM);
	}
}
void nnInsert(Array *A){
}
const float ex1[L1N]={-1,-1};
const float ex2[L1N]={-1,+1};
const float ex3[L1N]={+1,-1};
const float ex4[L1N]={+1,+1};
int main(){
	int i,j,k;
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	net->size=LAYERS;
	net->L[0]=(Layer *)malloc(sizeof(Layer));
	net->L[0]->in=(Array *)malloc(sizeof(Array));
	net->L[0]->out=(Array *)malloc(sizeof(Array));
	net->L[0]->in->len=L1N;
	net->L[0]->in->el=(float *)malloc(L1N*sizeof(float));
	net->L[0]->out->len=L1M;
	net->L[0]->out->el=(float *)malloc(L1M*sizeof(float));
	for(i=0;i<LAYERS;i++){
		if(i>0){
			net->L[i]=(Layer *)malloc(sizeof(Layer));
			net->L[i]->in=net->L[i-1]->out;
			net->L[i]->out=(Array *)malloc(sizeof(Array));
			net->L[i]->in->len=nDim[i];
			net->L[i]->in->el=(float *)malloc(nDim[i]*sizeof(float));
			net->L[i]->out->len=mDim[i];
			net->L[i]->out->el=(float *)malloc(mDim[i]*sizeof(float));
		}
//		net->L[i]=(Layer *)malloc(sizeof(Layer));
		net->L[i]->M=(Matrix *)malloc(sizeof(Matrix));
		memcpy(&net->L[i]->M->height,&nDim[i],sizeof(int));
		memcpy(&net->L[i]->M->width,&mDim[i],sizeof(int));
		memcpy(&net->L[i]->M->stride,&mDim[i],sizeof(int));
		net->L[i]->M->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
	}
	Matrix *pM=net->L[0]->M;
	PRINTMATRIX(net->L[0]->M);
	nnRand(net);
	PRINTMATRIX(net->L[0]->M);
	Array *pA=net->L[0]->in;
	PRINTARRAY(pA);
	randArray(pA);
	PRINTARRAY(pA);

	//pA=&ex1;
	//nnInsert(pA);

	Array *py=net->L[0]->out;
	PRINTARRAY(py);
	MatMul(*pM,*pA,*py);
	PRINTARRAY(py);
}
