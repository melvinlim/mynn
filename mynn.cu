#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "manip.cu"
#include "matrixmul.cu"

#define L1M 2
#define L1N 40
#define L2M 40
#define L2N 2

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};

void nnInsert(Net *N,Array *x){
	memcpy(N->L[0]->in->el,x->el,x->len*sizeof(float));
}
/*
Array *nnForward(Net *N){
	int i;
	for(i=0;i<LAYERS;i++){
//printf("***********%d\n",i);
//		PRINTARRAY(N->L[i]->in);
//		PRINTARRAY(N->L[i]->out);
		MatMul(*N->L[i]->M,*N->L[i]->in,*N->L[i]->out,*N->L[i]->deriv);
//		PRINTARRAY(N->L[i]->in);
//		PRINTARRAY(N->L[i]->out);
	}
	return N->L[LAYERS-1]->out;
}
*/
void nnError(Array *err,const Array *y0,const Array *yTarget){
	int i;
	int n=y0->len;
	for(i=0;i<n;i++){
		err->el[i]=(y0->el[i]-yTarget->el[i]);
	}
}
float nnTotalError(const Array *y0,const Array *y){
	int i;
	int n=y0->len;
	float ret=0;
	for(i=0;i<n;i++){
		ret+=fabs(y0->el[i]-y->el[i]);
		ret*=ret;
	}
	return(ret/2.0);
}
const float ex1[L1M]={-1,-1};
const float ex2[L1M]={-1,+1};
const float ex3[L1M]={+1,-1};
const float ex4[L1M]={+1,+1};
//const float ans1[L2N]={-1,+1};
//const float ans2[L2N]={+1,-1};
//const float ans3[L2N]={+1,-1};
//const float ans4[L2N]={-1,+1};
const float ans1[L2N]={-1,+1};
const float ans2[L2N]={+1,-1};
const float ans3[L2N]={+1,-1};
const float ans4[L2N]={+1,-1};
int main(){
	int i;
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	net->size=LAYERS;
	for(i=0;i<LAYERS;i++){
		net->L[i]=(Layer *)malloc(sizeof(Layer));
		if(i>0){
			net->L[i]->in=net->L[i-1]->out;
		}else{
			net->L[0]->in=(Array *)malloc(sizeof(Array));
		}
		net->L[i]->out=(Array *)malloc(sizeof(Array));
		net->L[i]->deriv=(Array *)malloc(sizeof(Array));
		net->L[i]->delta=(Array *)malloc(sizeof(Array));
		net->L[i]->in->len=mDim[i];
		net->L[i]->in->el=(float *)malloc(mDim[i]*sizeof(float));
		net->L[i]->out->len=nDim[i];
		net->L[i]->out->el=(float *)malloc(nDim[i]*sizeof(float));
		net->L[i]->deriv->len=nDim[i];
		net->L[i]->deriv->el=(float *)malloc(nDim[i]*sizeof(float));
		net->L[i]->delta->len=nDim[i];
		net->L[i]->delta->el=(float *)malloc(nDim[i]*sizeof(float));

		net->L[i]->M=(Matrix *)malloc(sizeof(Matrix));
		net->L[i]->M->height=nDim[i];
		net->L[i]->M->width=mDim[i];
		net->L[i]->M->stride=mDim[i];
		net->L[i]->M->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
		net->L[i]->dW=(Matrix *)malloc(sizeof(Matrix));
		net->L[i]->dW->height=nDim[i];
		net->L[i]->dW->width=mDim[i];
		net->L[i]->dW->stride=mDim[i];
		net->L[i]->dW->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
	}

	nnRand(net);
	for(i=0;i<LAYERS;i++){
		PRINTMATRIX(net->L[i]->M);
	}

	Array *p1,*p2,*p3,*p4,*ret;
	p1=CREATEARRAY(ex1,L1M);
	p2=CREATEARRAY(ex2,L1M);
	p3=CREATEARRAY(ex3,L1M);
	p4=CREATEARRAY(ex4,L1M);
	Array *pAns1,*pAns2,*pAns3,*pAns4;
	pAns1=CREATEARRAY(ans1,L2N);
	pAns2=CREATEARRAY(ans2,L2N);
	pAns3=CREATEARRAY(ans3,L2N);
	pAns4=CREATEARRAY(ans4,L2N);

	Array *pError;
	pError=CREATEARRAY(ans4,L2N);

	ret=CREATEARRAY(0,L2N);

	nnInsert(net,p1);
	ret=nnForward(net);
	PRINTARRAY(ret);
	nnError(pError,ret,pAns1);
	float err=nnTotalError(ret,pAns1);
	printf("err:%f\n",err);
	nnBackProp(net,pError);


	Array **pInputs=(Array **)malloc(4*sizeof(Array *));
	pInputs[0]=p1;
	pInputs[1]=p2;
	pInputs[2]=p3;
	pInputs[3]=p4;
	Array **pOutputs=(Array **)malloc(4*sizeof(Array *));
	pOutputs[0]=pAns1;
	pOutputs[1]=pAns2;
	pOutputs[2]=pAns3;
	pOutputs[3]=pAns4;
	Array *pIn,*pOut;
	int tmpvar;
	for(i=0;i<1000;i++){
		tmpvar=rand()%4;
		pIn=pInputs[tmpvar];
		pOut=pOutputs[tmpvar];
		nnInsert(net,pIn);
		ret=nnForward(net);
		nnError(pError,ret,pOut);
		err=nnTotalError(ret,pOut);
		printf("out:[%f,%f] targ:[%f,%f] err:%f\n",net->L[LAYERS-1]->out->el[0],net->L[LAYERS-1]->out->el[1],pOut->el[0],pOut->el[1],err);
		nnBackProp(net,pError);
	}

	for(i=0;i<LAYERS;i++){
		PRINTARRAY(net->L[i]->out);
	}
}
