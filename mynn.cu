#ifndef _MYNN
#define _MYNN

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <vector>

using namespace std;

#include "types.h"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (NINPUTS)
#define L1N (7)
#define L2M (7)
#define L2N (NOUTPUTS)

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};

void PRINTINFO(Array<double> *pIn,Array<double> *answer,Array<double> *pOut,Array<double> *pErr){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:[%f,%f]\n",
	pIn->el[0],pIn->el[1],
	answer->el[0],answer->el[1],
	pOut->el[0],pOut->el[1],
	pErr->el[0],pErr->el[1]
	);
}
double ex1[NINPUTS]={-1,-1};
double ex2[NINPUTS]={-1,+1};
double ex3[NINPUTS]={+1,-1};
double ex4[NINPUTS]={+1,+1};
double ans1[NOUTPUTS]={-1,+1};
double ans2[NOUTPUTS]={+1,-1};
double ans3[NOUTPUTS]={+1,-1};
double ans4[NOUTPUTS]={-1,+1};
//const double ans1[NOUTPUTS]={-1,+1};
//const double ans2[NOUTPUTS]={+1,-1};
//const double ans3[NOUTPUTS]={+1,-1};
//const double ans4[NOUTPUTS]={+1,-1};
int main(){
	int i;
	srand(time(0));
	Net *net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,nDim[i],mDim[i]);
	}

	net->print();
	net->rand();
	net->print();
/*
	Array<double> *p1,*p2,*p3,*p4;
	Array<double> *pAns1,*pAns2,*pAns3,*pAns4;

	p1=new Array<double>(ex1,NINPUTS);
	p2=new Array<double>(ex2,NINPUTS);
	p3=new Array<double>(ex3,NINPUTS);
	p4=new Array<double>(ex4,NINPUTS);

	pAns1=new Array<double>(ans1,NOUTPUTS);
	pAns2=new Array<double>(ans2,NOUTPUTS);
	pAns3=new Array<double>(ans3,NOUTPUTS);
	pAns4=new Array<double>(ans4,NOUTPUTS);
*/
	double **pInputs=(double **)malloc(4*sizeof(double *));
	pInputs[0]=ex1;
	pInputs[1]=ex2;
	pInputs[2]=ex3;
	pInputs[3]=ex4;
	double **pOutputs=(double **)malloc(4*sizeof(double *));
	pOutputs[0]=ans1;
	pOutputs[1]=ans2;
	pOutputs[2]=ans3;
	pOutputs[3]=ans4;
	Array<double> *pIn,*pOut;
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=rand()%4;
		pIn=new Array<double>(pInputs[tmpvar],NINPUTS);
		pOut=new Array<double>(pOutputs[tmpvar],NOUTPUTS);
		net->train(pIn,pOut);
		PRINTINFO(pIn,net->answer,pOut,net->error);
		delete pIn;
		delete pOut;
	}
}

#endif
