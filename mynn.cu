#ifndef _MYNN
#define _MYNN

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "types.h"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (NINPUTS)
#define L1N (7)
#define L2M (7)
#define L2N (NOUTPUTS)

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};

void PRINTINFO(Array *pIn,Array *answer,Array *pOut,Array *pErr){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:[%f,%f]\n",
	pIn->el[0],pIn->el[1],
	answer->el[0],answer->el[1],
	pOut->el[0],pOut->el[1],
	pErr->el[0],pErr->el[1]
	);
}
const float ex1[NINPUTS]={-1,-1};
const float ex2[NINPUTS]={-1,+1};
const float ex3[NINPUTS]={+1,-1};
const float ex4[NINPUTS]={+1,+1};
const float ans1[NOUTPUTS]={-1,+1};
const float ans2[NOUTPUTS]={+1,-1};
const float ans3[NOUTPUTS]={+1,-1};
const float ans4[NOUTPUTS]={-1,+1};
//const float ans1[NOUTPUTS]={-1,+1};
//const float ans2[NOUTPUTS]={+1,-1};
//const float ans3[NOUTPUTS]={+1,-1};
//const float ans4[NOUTPUTS]={+1,-1};
int main(){
	int i;
	srand(time(0));
	Net *net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,nDim[i],mDim[i]);
	}

	net->print();
	net->rand();
	net->print();

	Array *p1,*p2,*p3,*p4;
	Array *pAns1,*pAns2,*pAns3,*pAns4;

	p1=new Array(ex1,NINPUTS);
	p2=new Array(ex2,NINPUTS);
	p3=new Array(ex3,NINPUTS);
	p4=new Array(ex4,NINPUTS);

	pAns1=new Array(ans1,NOUTPUTS);
	pAns2=new Array(ans2,NOUTPUTS);
	pAns3=new Array(ans3,NOUTPUTS);
	pAns4=new Array(ans4,NOUTPUTS);

	Array **pInputs=(Array **)malloc(4*sizeof(Array *));
	pInputs[0]=p1;
	pInputs[1]=p2;
	pInputs[2]=p3;
	pInputs[3]=p4;
	Array **pOutputs=(Array **)malloc(4*sizeof(Array *));
	pOutputs[0]=pAns1;
	pOutputs[1]=pAns2;
	pOutputs[2]=pAns3;
	pOutputs[3]=pAns4;
	Array *pIn,*pOut;
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=rand()%4;
		pIn=pInputs[tmpvar];
		pOut=pOutputs[tmpvar];
		net->train(pIn,pOut);
		PRINTINFO(pIn,net->answer,pOut,net->error);
	}
}

#endif
