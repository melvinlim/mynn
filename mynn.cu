#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "manip.cu"
#include "matrixmul.cu"

#define L1N 40
#define L1M 2
#define L2N 2
#define L2M 40

#define RANDSCALING 10	//scale random weights to be from -0.1 to +0.1

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
/*
void PRINTMATRIX(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			printf("[%i,%i]%.09f ",i,j,M->elements[i*M->stride+j]);
		}
		printf("\n");
	}
	printf("\n");
}
void PRINTARRAY(Array *A){
	int i;
	float *x;
	int sz=A->len;
	x=A->el;
	for(i=0;i<sz;i++){
		printf("[%i]%.02f\t",i,*x++);
	}
	printf("\n");
}
void randMatrix(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			M->elements[i*M->stride+j]=
			(random()-(RAND_MAX/2))*2.0/((float)RAND_MAX)/((float)RANDSCALING);
		}
	}
}
void randArray(Array *A){
	int i;
	for(i=0;i<A->len;i++){
		A->el[i]=
		(random()-(RAND_MAX/2))*2.0/((float)RAND_MAX)/((float)RANDSCALING);
	}
}
void nnRand(Net *N){
	Matrix *pM;
	int i;
	int n=N->size;
	for(i=0;i<n;i++){
		pM=N->L[i]->M;
		randMatrix(pM);
	}
}
Array *CREATEARRAY(const float *x,int n){
	Array *p=(Array *)malloc(sizeof(Array));
	p->len=n;
	p->el=(float *)malloc(n*sizeof(float));
	if(x){
		memcpy(p->el,x,n*sizeof(float));
	}
	return(p);
}
*/
void nnInsert(Net *N,Array *x){
	memcpy(N->L[0]->in->el,x->el,x->len*sizeof(float));
	N->L[0]->in->len=x->len;	
}
Array *nnForward(Net *N){
	int i;
	for(i=0;i<LAYERS;i++){
//printf("***********%d\n",i);
//		PRINTARRAY(N->L[i]->in);
//		PRINTARRAY(N->L[i]->out);
		MatMul(*N->L[i]->M,*N->L[i]->in,*N->L[i]->out,*N->L[i]->deriv);
//		PRINTARRAY(N->L[i]->in);
//		PRINTARRAY(N->L[i]->out);
	}
	return N->L[LAYERS-1]->out;
}
void nnError(Array *err,const Array *y0,const Array *yTarget){
	int i;
	int n=y0->len;
	float ret=0;
	for(i=0;i<n;i++){
		err->el[i]=(y0->el[i]-yTarget->el[i]);
	}
}
float nnTotalError(const Array *y0,const Array *y){
	int i;
	int n=y0->len;
	float ret=0;
	for(i=0;i<n;i++){
		ret+=fabs(y0->el[i]-y->el[i]);
		ret*=ret;
	}
	return(ret/2.0);
}
const float ex1[L1M]={-1,-1};
const float ex2[L1M]={-1,+1};
const float ex3[L1M]={+1,-1};
const float ex4[L1M]={+1,+1};
//const float ans1[L2N]={-1,+1};
//const float ans2[L2N]={+1,-1};
//const float ans3[L2N]={+1,-1};
//const float ans4[L2N]={-1,+1};
const float ans1[L2N]={-1,+1};
const float ans2[L2N]={-1,+1};
const float ans3[L2N]={-1,+1};
const float ans4[L2N]={+1,-1};
int main(){
	int i,j,k;
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	net->size=LAYERS;
	net->L[0]=(Layer *)malloc(sizeof(Layer));
	net->L[0]->in=(Array *)malloc(sizeof(Array));
	net->L[0]->out=(Array *)malloc(sizeof(Array));
	net->L[0]->deriv=(Array *)malloc(sizeof(Array));
	net->L[0]->in->len=L1M;
	net->L[0]->in->el=(float *)malloc(L1M*sizeof(float));
	net->L[0]->out->len=L1N;
	net->L[0]->out->el=(float *)malloc(L1N*sizeof(float));
	net->L[0]->deriv->len=L1N;
	net->L[0]->deriv->el=(float *)malloc(L1N*sizeof(float));
	for(i=0;i<LAYERS;i++){
		if(i>0){
			net->L[i]=(Layer *)malloc(sizeof(Layer));
			net->L[i]->in=net->L[i-1]->out;
			net->L[i]->out=(Array *)malloc(sizeof(Array));
			net->L[i]->deriv=(Array *)malloc(sizeof(Array));
			net->L[i]->in->len=mDim[i];
			net->L[i]->in->el=(float *)malloc(mDim[i]*sizeof(float));
			net->L[i]->out->len=nDim[i];
			net->L[i]->out->el=(float *)malloc(nDim[i]*sizeof(float));
			net->L[i]->deriv->len=nDim[i];
			net->L[i]->deriv->el=(float *)malloc(nDim[i]*sizeof(float));
		}
		net->L[i]->M=(Matrix *)malloc(sizeof(Matrix));
		net->L[i]->M->height=nDim[i];
		net->L[i]->M->width=mDim[i];
		net->L[i]->M->stride=mDim[i];
		net->L[i]->M->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
		net->L[i]->dW=(Matrix *)malloc(sizeof(Matrix));
		net->L[i]->dW->height=nDim[i];
		net->L[i]->dW->width=mDim[i];
		net->L[i]->dW->stride=mDim[i];
printf("i=%d,ndim=%d %d\n",i,nDim[i],net->L[i]->M->height);
		net->L[i]->dW->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
	}
	//Matrix *pM=net->L[0]->M;
	//PRINTMATRIX(net->L[0]->M);
	nnRand(net);
	for(i=0;i<LAYERS;i++){
		PRINTMATRIX(net->L[i]->M);
	}
	//PRINTMATRIX(net->L[0]->M);

	Array *p1,*p2,*p3,*p4,*ret;
	p1=CREATEARRAY(ex1,L1M);
	p2=CREATEARRAY(ex2,L1M);
	p3=CREATEARRAY(ex3,L1M);
	p4=CREATEARRAY(ex4,L1M);
	Array *pAns1,*pAns2,*pAns3,*pAns4;
	pAns1=CREATEARRAY(ans1,L2N);
	pAns2=CREATEARRAY(ans2,L2N);
	pAns3=CREATEARRAY(ans3,L2N);
	pAns4=CREATEARRAY(ans4,L2N);

	Array *pError;
	pError=CREATEARRAY(ans4,L2N);

	ret=CREATEARRAY(0,L2N);

	nnInsert(net,p1);
	ret=nnForward(net);
	PRINTARRAY(ret);

	nnError(pError,ret,pAns1);
	float err=nnTotalError(ret,pAns1);
	printf("err:%f\n",err);
	nnBackProp(net,pError);


	Array **pInputs=(Array **)malloc(4*sizeof(Array *));
	pInputs[0]=p1;
	pInputs[1]=p2;
	pInputs[2]=p3;
	pInputs[3]=p4;
	Array **pOutputs=(Array **)malloc(4*sizeof(Array *));
	pOutputs[0]=pAns1;
	pOutputs[1]=pAns2;
	pOutputs[2]=pAns3;
	pOutputs[3]=pAns4;
	Array *pIn,*pOut;
	int tmpvar;
	for(i=0;i<1000;i++){
		tmpvar=rand()%4;
		pIn=pInputs[tmpvar];
		pOut=pOutputs[tmpvar];
		nnInsert(net,pIn);
		ret=nnForward(net);
		nnError(pError,ret,pOut);
		err=nnTotalError(ret,pOut);
		printf("out:[%f,%f] targ:[%f,%f] err:%f\n",net->L[LAYERS-1]->out->el[0],net->L[LAYERS-1]->out->el[1],pOut->el[0],pOut->el[1],err);
		nnBackProp(net,pError);
	}

	for(i=0;i<LAYERS;i++){
		PRINTARRAY(net->L[i]->out);
	}
}
