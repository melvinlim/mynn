#ifndef _MYNN
#define _MYNN

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <vector>

using namespace std;

#include "types.h"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (NINPUTS)
#define L1N (7)
#define L2M (7)
#define L2N (NOUTPUTS)

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};

void PRINTINFO(Array<float> *pIn,Array<float> *answer,Array<float> *pOut,Array<float> *pErr){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:[%f,%f]\n",
	pIn->el[0],pIn->el[1],
	answer->el[0],answer->el[1],
	pOut->el[0],pOut->el[1],
	pErr->el[0],pErr->el[1]
	);
}
float ex1[NINPUTS]={-1,-1};
float ex2[NINPUTS]={-1,+1};
float ex3[NINPUTS]={+1,-1};
float ex4[NINPUTS]={+1,+1};
float ans1[NOUTPUTS]={-1,+1};
float ans2[NOUTPUTS]={+1,-1};
float ans3[NOUTPUTS]={+1,-1};
float ans4[NOUTPUTS]={-1,+1};
//const float ans1[NOUTPUTS]={-1,+1};
//const float ans2[NOUTPUTS]={+1,-1};
//const float ans3[NOUTPUTS]={+1,-1};
//const float ans4[NOUTPUTS]={+1,-1};
int main(){
	int i;
	srand(time(0));
	Net *net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,nDim[i],mDim[i]);
	}

	net->print();
	net->rand();
	net->print();
/*
	Array<float> *p1,*p2,*p3,*p4;
	Array<float> *pAns1,*pAns2,*pAns3,*pAns4;

	p1=new Array<float>(ex1,NINPUTS);
	p2=new Array<float>(ex2,NINPUTS);
	p3=new Array<float>(ex3,NINPUTS);
	p4=new Array<float>(ex4,NINPUTS);

	pAns1=new Array<float>(ans1,NOUTPUTS);
	pAns2=new Array<float>(ans2,NOUTPUTS);
	pAns3=new Array<float>(ans3,NOUTPUTS);
	pAns4=new Array<float>(ans4,NOUTPUTS);
*/
	float **pInputs=(float **)malloc(4*sizeof(float *));
	pInputs[0]=ex1;
	pInputs[1]=ex2;
	pInputs[2]=ex3;
	pInputs[3]=ex4;
	float **pOutputs=(float **)malloc(4*sizeof(float *));
	pOutputs[0]=ans1;
	pOutputs[1]=ans2;
	pOutputs[2]=ans3;
	pOutputs[3]=ans4;
	Array<float> *pIn,*pOut;
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=rand()%4;
		pIn=new Array<float>(pInputs[tmpvar],NINPUTS);
		pOut=new Array<float>(pOutputs[tmpvar],NOUTPUTS);
		net->train(pIn,pOut);
		PRINTINFO(pIn,net->answer,pOut,net->error);
		delete pIn;
		delete pOut;
	}
}

#endif
