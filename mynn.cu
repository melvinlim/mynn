#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#include "matrixmul.cu"

#define LAYERS 3
#define L1N 6
#define L1M 6
#define L2N 6
#define L2M 40
#define L3N 40
#define L3M 40

const int nDim[LAYERS]={L1N,L2N,L3N};
const int mDim[LAYERS]={L1M,L2M,L3M};

struct Layer{
	Array *in;
	Matrix *M;
	Array *out;
};
struct Net{
	Layer **L;
};
void PRINTMATRIX(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			printf("[%i,%i]%.02f\t",i,j,M->elements[i*M->stride+j]);
		}
	}
	printf("\n");
}
void PRINTARRAY(Array *A){
	int i;
	float *x;
	int sz=A->len;
	x=A->el;
	for(i=0;i<sz;i++){
		printf("[%i]%.02f\t",i,*x++);
	}
	printf("\n");
}
int main(){
	int i,j,k;
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	net->L[0]=(Layer *)malloc(sizeof(Layer));
	net->L[0]->in=(Array *)malloc(sizeof(Array));
	net->L[0]->out=(Array *)malloc(sizeof(Array));
	net->L[0]->in->len=L1N;
	net->L[0]->in->el=(float *)malloc(L1N*sizeof(float));
	net->L[0]->out->len=L1M;
	net->L[0]->out->el=(float *)malloc(L1M*sizeof(float));
	for(i=0;i<LAYERS;i++){
		if(i>0){
			net->L[i]=(Layer *)malloc(sizeof(Layer));
			net->L[i]->in=net->L[i-1]->out;
			net->L[i]->out=(Array *)malloc(sizeof(Array));
			net->L[i]->in->len=nDim[i];
			net->L[i]->in->el=(float *)malloc(nDim[i]*sizeof(float));
			net->L[i]->out->len=mDim[i];
			net->L[i]->out->el=(float *)malloc(mDim[i]*sizeof(float));
		}
//		net->L[i]=(Layer *)malloc(sizeof(Layer));
		net->L[i]->M=(Matrix *)malloc(sizeof(Matrix));
		memcpy(&net->L[i]->M->height,&nDim[i],sizeof(int));
		memcpy(&net->L[i]->M->width,&mDim[i],sizeof(int));
		memcpy(&net->L[i]->M->stride,&mDim[i],sizeof(int));
		net->L[i]->M->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
	}
	PRINTMATRIX(net->L[0]->M);
	k=0;
	for(i=0;i<net->L[0]->M->height;i++){
		for(j=0;j<net->L[0]->M->width;j++){
			net->L[0]->M->elements[i*net->L[0]->M->stride+j]=k++;
		}
	}
	PRINTMATRIX(net->L[0]->M);
	Array *pA=net->L[0]->in;
	PRINTARRAY(pA);
	for(i=0;i<pA->len;i++){
		pA->el[i]=i;
	}
	PRINTARRAY(pA);
	Matrix *Mptr=net->L[0]->M;
	//MatMul requires matrices to be multiples of BLOCK_SIZE (declared in matmul.cu) and possibly to be square.
	MatMul(*Mptr,*Mptr,*Mptr);
	PRINTMATRIX(net->L[0]->M);
	Array *py=net->L[0]->out;
	PRINTARRAY(py);
	MatMul(*Mptr,*pA,*py);
	PRINTARRAY(py);
}
