#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "manip.cu"
#include "matrixmul.cu"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (NINPUTS)
#define L1N (7)
#define L2M (7)
#define L2N (NOUTPUTS)

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};

void PRINTINFO(Array *pIn,Net *net,Array *pOut,float err){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:%f\n",
	pIn->el[0],pIn->el[1],
	net->L[LAYERS-1]->out->el[0],net->L[LAYERS-1]->out->el[1],
	pOut->el[0],pOut->el[1],err);
}
void nnError(Array *err,const Array *y0,const Array *yTarget){
	int i;
	for(i=0;i<y0->n;i++){
		err->el[i]=(y0->el[i]-yTarget->el[i]);
	}
}
float nnTotalError(const Array *y0,const Array *y){
	int i;
	int n=y0->n;
	float ret=0;
	for(i=0;i<n;i++){
		ret+=fabs(y0->el[i]-y->el[i]);
		ret*=ret;
	}
	return(ret/2.0);
}
const float ex1[NINPUTS]={-1,-1};
const float ex2[NINPUTS]={-1,+1};
const float ex3[NINPUTS]={+1,-1};
const float ex4[NINPUTS]={+1,+1};
const float ans1[NOUTPUTS]={-1,+1};
const float ans2[NOUTPUTS]={+1,-1};
const float ans3[NOUTPUTS]={+1,-1};
const float ans4[NOUTPUTS]={-1,+1};
//const float ans1[NOUTPUTS]={-1,+1};
//const float ans2[NOUTPUTS]={+1,-1};
//const float ans3[NOUTPUTS]={+1,-1};
//const float ans4[NOUTPUTS]={+1,-1};
int main(){
	int i;
	srand(time(0));
	Net *net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,nDim[i],mDim[i]);
	}

	nnRand(net);
	for(i=0;i<LAYERS;i++){
		PRINTMATRIX(net->L[i]->M);
	}

	Array *p1,*p2,*p3,*p4,*ret;
	Array *pAns1,*pAns2,*pAns3,*pAns4;
	Array *pError;

	p1=new Array(ex1,NINPUTS);
	p2=new Array(ex2,NINPUTS);
	p3=new Array(ex3,NINPUTS);
	p4=new Array(ex4,NINPUTS);

	pAns1=new Array(ans1,NOUTPUTS);
	pAns2=new Array(ans2,NOUTPUTS);
	pAns3=new Array(ans3,NOUTPUTS);
	pAns4=new Array(ans4,NOUTPUTS);

	pError=new Array(0,NOUTPUTS);

	ret=net->input(p1);
	PRINTARRAY(ret);

	nnError(pError,ret,pAns1);
	float err=nnTotalError(ret,pAns1);

	printf("err:%f\n",err);

	net->insertError(p1,pError);

	Array **pInputs=(Array **)malloc(4*sizeof(Array *));
	pInputs[0]=p1;
	pInputs[1]=p2;
	pInputs[2]=p3;
	pInputs[3]=p4;
	Array **pOutputs=(Array **)malloc(4*sizeof(Array *));
	pOutputs[0]=pAns1;
	pOutputs[1]=pAns2;
	pOutputs[2]=pAns3;
	pOutputs[3]=pAns4;
	Array *pIn,*pOut;
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=rand()%4;
		pIn=pInputs[tmpvar];
		pOut=pOutputs[tmpvar];
		net->input(pIn);
		ret=net->input(pIn);
		nnError(pError,ret,pOut);
		err=nnTotalError(ret,pOut);
		PRINTINFO(pIn,net,pOut,err);
		net->insertError(pIn,pError);
	}
}
