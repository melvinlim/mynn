#ifndef _MYNN
#define _MYNN

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <vector>

using namespace std;

#include "types.h"
#include "libnn.h"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (7)
#define L1N (NINPUTS)
#define L2M (NOUTPUTS)
#define L2N (7)

const int mDim[LAYERS]={L1M,L2M};//,L3M};
const int nDim[LAYERS]={L1N,L2N};//,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};

void PRINTINFO(const Array<double> &pIn,const Array<double> &answer,const Array<double> &pOut,const Array<double> &pErr){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:[%f,%f]\n",
	pIn(0),pIn(1),
	answer(0),answer(1),
	pOut(0),pOut(1),
	pErr(0),pErr(1)
	);
}
double ex1[NINPUTS]={-1,-1};
double ex2[NINPUTS]={-1,+1};
double ex3[NINPUTS]={+1,-1};
double ex4[NINPUTS]={+1,+1};
double ans1[NOUTPUTS]={-1,+1};
double ans2[NOUTPUTS]={+1,-1};
double ans3[NOUTPUTS]={+1,-1};
double ans4[NOUTPUTS]={-1,+1};
//const double ans1[NOUTPUTS]={-1,+1};
//const double ans2[NOUTPUTS]={+1,-1};
//const double ans3[NOUTPUTS]={+1,-1};
//const double ans4[NOUTPUTS]={+1,-1};
int main(){
	int i,j;
	srand(time(0));
	Net *net=0;
	net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,mDim[i],nDim[i]);
	}

	net->rand();

	Matrix<double> mat1=Matrix<double>(10,4);

	Array<double> arr1=Array<double>(10);
	Array<double> arr2=Array<double>();
	arr2.resize(10);
	for(i=0;i<10;i++){
		arr1(i)=i;
		arr2(i)=i+2;
		for(j=0;j<4;j++){
			mat1(i,j)=(i+1)*(j+2);
		}
	}
	for(i=0;i<10;i++){
		for(j=0;j<4;j++){
			assert(mat1(i,j)==(i+1)*(j+2));
		}
	}
	arr1.print();
	arr2.print();
	arr1=arr1+arr2;
	arr1.print();
	arr1[0]+=5;
	arr1[5]+=15;
	arr1(2)=arr1[3]*4;
	arr1.print();
//	return 0;

	vector<Array<double> > pIn,pOut;
	pIn.resize(4);
	pIn[0]=Array<double>(ex1,NINPUTS);
	pIn[1]=Array<double>(ex2,NINPUTS);
	pIn[2]=Array<double>(ex3,NINPUTS);
	pIn[3]=Array<double>(ex4,NINPUTS);
	for(Array<double> x:pIn)
		x.print();
	pOut.resize(4);
	pOut[0]=Array<double>(ans1,NOUTPUTS);
	pOut[1]=Array<double>(ans2,NOUTPUTS);
	pOut[2]=Array<double>(ans3,NOUTPUTS);
	pOut[3]=Array<double>(ans4,NOUTPUTS);
	for(Array<double> x:pOut)
		x.print();
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=i%4;
		net->train(pIn[tmpvar],pOut[tmpvar]);
		PRINTINFO(pIn[tmpvar],net->answer,pOut[tmpvar],net->error);
	}
}

#endif
