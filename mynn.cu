#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define LAYERS 3
#define L1N 40
#define L1M 40
#define L2N 40
#define L2M 40
#define L3N 40
#define L3M 40

const int nDim[LAYERS]={L1N,L2N,L3N};
const int mDim[LAYERS]={L1M,L2M,L3M};

struct Layer{
	float **L;
	int *N;
	int *M;
};
struct Net{
	Layer **L;
};
int main(){
	int i,j;
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	for(i=0;i<LAYERS;i++){
		net->L[i]=(Layer *)malloc(sizeof(Layer));
		net->L[i]->N=(int *)malloc(sizeof(int));
		net->L[i]->M=(int *)malloc(sizeof(int));
		memcpy(net->L[i]->N,&nDim[i],sizeof(int));
		memcpy(net->L[i]->M,&mDim[i],sizeof(int));
		net->L[i]->L=(float **)malloc(nDim[i]*sizeof(float *));
		for(j=0;j<nDim[i];j++){
			net->L[i]->L[j]=(float *)malloc(mDim[i]*sizeof(float));
		}
	}
}
