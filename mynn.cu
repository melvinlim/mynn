#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#include "matrixmul.cu"

#define LAYERS 3
#define L1N 6
#define L1M 6
#define L2N 40
#define L2M 40
#define L3N 40
#define L3M 40

const int nDim[LAYERS]={L1N,L2N,L3N};
const int mDim[LAYERS]={L1M,L2M,L3M};

struct Layer{
	Matrix *M;
};
struct Net{
	Layer **L;
};
void PRINTMATRIX(Matrix *M){
	int i,j;
	for(i=0;i<M->height;i++){
		for(j=0;j<M->width;j++){
			printf("[%i,%i]%.02f\t",i,j,M->elements[i*M->stride+j]);
		}
	}
	printf("\n");
}
void PRINTARRAY(float *x,int sz){
	int i;
	for(i=0;i<sz;i++){
		printf("[%i]%.02f\t",i,*x++);
	}
	printf("\n");
}
int main(){
	int i,j,k;
	float test[L1N*L1M];
	Net *net;
	net=(Net *)malloc(sizeof(Net));
	net->L=(Layer **)malloc(LAYERS*sizeof(Layer *));
	for(i=0;i<LAYERS;i++){
		net->L[i]=(Layer *)malloc(sizeof(Layer));
		net->L[i]->M=(Matrix *)malloc(sizeof(Matrix));
		memcpy(&net->L[i]->M->height,&nDim[i],sizeof(int));
		memcpy(&net->L[i]->M->width,&mDim[i],sizeof(int));
		memcpy(&net->L[i]->M->stride,&mDim[i],sizeof(int));
		net->L[i]->M->elements=(float *)malloc(nDim[i]*mDim[i]*sizeof(float));
	}
	for(i=0;i<L1N*L1M;i++){
		test[i]=i;
	}
	PRINTARRAY(test,L1N*L1M);
	PRINTMATRIX(net->L[0]->M);
	//memcpy(net->L[0]->L,test,L1N*L1M*sizeof(float));
	k=0;
	for(i=0;i<net->L[0]->M->height;i++){
		for(j=0;j<net->L[0]->M->width;j++){
			net->L[0]->M->elements[i*net->L[0]->M->stride+j]=k++;
		}
	}
	PRINTMATRIX(net->L[0]->M);
	Matrix *Mptr=net->L[0]->M;
	//MatMul requires matrices to be multiples of BLOCK_SIZE (declared in matmul.cu) and possibly to be square.
	MatMul(*Mptr,*Mptr,*Mptr);
	PRINTMATRIX(net->L[0]->M);
}
