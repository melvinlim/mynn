#ifndef _MYNN
#define _MYNN

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <vector>

using namespace std;

#include "types.h"

#define NINPUTS (2)
#define NOUTPUTS (2)

#define L1M (NINPUTS)
#define L1N (7)
#define L2M (7)
#define L2N (NOUTPUTS)

const int nDim[LAYERS]={L1N,L2N};//,L3N};
const int mDim[LAYERS]={L1M,L2M};//,L3M};
//const int nDim[LAYERS]={L1N,L2N,L3N};
//const int mDim[LAYERS]={L1M,L2M,L3M};

void PRINTINFO(const Array<double> &pIn,const Array<double> &answer,const Array<double> &pOut,const Array<double> &pErr){
	printf("in:[%.0f,%.0f] out:[%f,%f] targ:[%.0f,%.0f] err:[%f,%f]\n",
	pIn(0),pIn(1),
	answer(0),answer(1),
	pOut(0),pOut(1),
	pErr(0),pErr(1)
	);
}
double ex1[NINPUTS]={-1,-1};
double ex2[NINPUTS]={-1,+1};
double ex3[NINPUTS]={+1,-1};
double ex4[NINPUTS]={+1,+1};
double ans1[NOUTPUTS]={-1,+1};
double ans2[NOUTPUTS]={+1,-1};
double ans3[NOUTPUTS]={+1,-1};
double ans4[NOUTPUTS]={-1,+1};
//const double ans1[NOUTPUTS]={-1,+1};
//const double ans2[NOUTPUTS]={+1,-1};
//const double ans3[NOUTPUTS]={+1,-1};
//const double ans4[NOUTPUTS]={+1,-1};
int main(){
	int i;
	srand(time(0));
	Net *net=new Net(LAYERS);
	for(i=0;i<LAYERS;i++){
		net->insertLayer(i,nDim[i],mDim[i]);
	}

	net->print();
	net->rand();
	net->print();
/*
	Array<double> *p1,*p2,*p3,*p4;
	Array<double> *pAns1,*pAns2,*pAns3,*pAns4;

	p1=new Array<double>(ex1,NINPUTS);
	p2=new Array<double>(ex2,NINPUTS);
	p3=new Array<double>(ex3,NINPUTS);
	p4=new Array<double>(ex4,NINPUTS);

	pAns1=new Array<double>(ans1,NOUTPUTS);
	pAns2=new Array<double>(ans2,NOUTPUTS);
	pAns3=new Array<double>(ans3,NOUTPUTS);
	pAns4=new Array<double>(ans4,NOUTPUTS);
*/

	vector<Array<double> > pIn,pOut;
	pIn.resize(4);
	pIn[0]=Array<double>(ex1,NINPUTS);
	pIn[1]=Array<double>(ex2,NINPUTS);
	pIn[2]=Array<double>(ex3,NINPUTS);
	pIn[3]=Array<double>(ex4,NINPUTS);
	for(Array<double> x:pIn)
		x.print();
	pOut.resize(4);
	pOut[0]=Array<double>(ans1,NOUTPUTS);
	pOut[1]=Array<double>(ans2,NOUTPUTS);
	pOut[2]=Array<double>(ans3,NOUTPUTS);
	pOut[3]=Array<double>(ans4,NOUTPUTS);
	for(Array<double> x:pOut)
		x.print();
//	pIn[i].print();
/*
	double **pInputs=(double **)malloc(4*sizeof(double *));
	pInputs[0]=ex1;
	pInputs[1]=ex2;
	pInputs[2]=ex3;
	pInputs[3]=ex4;
	double **pOutputs=(double **)malloc(4*sizeof(double *));
	pOutputs[0]=ans1;
	pOutputs[1]=ans2;
	pOutputs[2]=ans3;
	pOutputs[3]=ans4;
	Array<double> *pIn,*pOut;
*/
	int tmpvar;
	for(i=0;i<EPOCHS;i++){
		tmpvar=rand()%4;
//		pIn=new Array<double>(pInputs[tmpvar],NINPUTS);
//		pOut=new Array<double>(pOutputs[tmpvar],NOUTPUTS);
		net->train(pIn[tmpvar],pOut[tmpvar]);
		PRINTINFO(pIn[tmpvar],net->answer,pOut[tmpvar],net->error);
//		delete pIn;
//		delete pOut;
	}
}

#endif
