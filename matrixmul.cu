
#include <hip/hip_runtime.h>
#define LAYERS 2

#define GAMMA (0.1)

// Thread block size
//#define BLOCK_SIZE 16
#define BLOCK_SIZE 2

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
}Matrix;

typedef struct{
	int len;
	float *el;
}Array;

struct Layer{
	Array *in;
	Matrix *M;
	Matrix *dW;
	Array *out;
	Array *deriv;
};

struct Net{
	Layer **L;
	int size;
};

void updateWeights(Layer *L){
	int i,j;
	Matrix *A=L->M;
//printf("%d %d\n",A->width,A->height);
	Array *delta=L->deriv;
	Array *input=L->in;
	for(i=0;i<A->width;i++){
		for(j=0;j<A->height;j++){
			A->elements[j*A->width+i]-=GAMMA*input->el[i]*delta->el[j];
		}
	}
}

void bpDeltas(Layer *L1,Layer *L2){
	int j,k;
	float sum;
	Array *deriv=L1->deriv;
	Array *delta=L2->deriv;
	Matrix *W=L2->M;
	for(j=0;j<deriv->len;j++){
		sum=0;
		for(k=0;k<delta->len;k++){
			sum+=W->elements[j*W->width+k]*delta->el[k];
		}
		deriv->el[j]=deriv->el[j]*sum;
	}
}

void bpDeltas0(Layer *L,Array *error){
	int j;
	Array *deriv=L->deriv;
	for(j=0;j<deriv->len;j++){
		//storing delta in deriv...  should possibly have dedicated array...
		deriv->el[j]=deriv->el[j]*error->el[j];
	}
}

void nnBackProp(Net *N,Array *error){
	int i;
	bpDeltas0(N->L[LAYERS-1],error);
	for(i=LAYERS-2;i>=0;i--){
		bpDeltas(N->L[i],N->L[i+1]);
	}
	for(i=LAYERS-1;i>=0;i--){
		updateWeights(N->L[i]);
	}
}

//this actually does more than simply multiply.
__global__ void MatMulKernel(const Matrix A, const Array x, Array y, Array deriv){
	int i;
	float Cval=0;
	//int row = blockIdx.y * blockDim.y + threadIdx.y;
  //int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
	for(i=0;i<A.width;i++){
		Cval+=A.elements[row*A.width+i]*x.el[i];
	}
	float tmp=tanhf(Cval);
	y.el[row]=tmp;
	deriv.el[row]=1.0-(tmp*tmp);
}

void MatMul(const Matrix A, const Array x, Array y, Array deriv)
{
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);

		Array d_x;
		d_x.len=x.len;
		d_x.el=x.el;
		size_t xSz=x.len*sizeof(float);
    hipMalloc(&d_x.el,xSz);
    hipMemcpy(d_x.el,x.el,xSz,
	    hipMemcpyHostToDevice);
		
		Array d_y;
		d_y.len=y.len;
		d_y.el=y.el;
		size_t ySz=y.len*sizeof(float);
    hipMalloc(&d_y.el,ySz);
		
		Array d_deriv;
		d_deriv.len=deriv.len;
		d_deriv.el=deriv.el;
    hipMalloc(&d_deriv.el,ySz);

    // Invoke kernel
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    //dim3 dimGrid(dimBlock.x, A.height / dimBlock.y);
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, d_deriv);
    //MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y);

    // Read from device memory
    hipMemcpy(y.el, d_y.el, ySz,
    	hipMemcpyDeviceToHost);
    
		hipMemcpy(deriv.el, d_deriv.el, ySz,
    	hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_x.el);
    hipFree(d_y.el);
    hipFree(d_deriv.el);
}

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);

    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}
