#include "hip/hip_runtime.h"
void updateWeights(Layer *L){
	int i,j;
	Matrix *A=L->M;
//printf("%d %d\n",A->m,A->n);
	Array *delta=L->delta;
	//Array *input=L->in;
	for(j=0;j<A->n;j++){
		for(i=0;i<A->m;i++){
			//A->el[j*A->m+i]-=GAMMA*input->el[i]*delta->el[j];
		}
	}
}

void bpDeltas(Layer *L1,Layer *L2){
	int j,k;
	float sum;
	Array *deriv=L1->deriv;
	Array *delta1=L1->delta;
	Array *delta2=L2->delta;
	Matrix *W=L2->M;
	for(j=0;j<deriv->len;j++){
		sum=0;
		for(k=0;k<delta2->len;k++){
			sum+=W->el[j*W->m+k]*delta2->el[k];
		}
		delta1->el[j]=deriv->el[j]*sum;
	}
}

void bpDeltas0(Layer *L,const Array *error){
	int j;
	Array *deriv=L->deriv;
	Array *delta=L->delta;
	for(j=0;j<deriv->len;j++){
		delta->el[j]=deriv->el[j]*error->el[j];
		//delta->el[j]=error->el[j];
	}
}

void nnBackProp(Net *N,Array *error){
	int i;
	bpDeltas0(N->L[LAYERS-1],error);
	for(i=LAYERS-2;i>=0;i--){
		bpDeltas(N->L[i],N->L[i+1]);
	}
	for(i=LAYERS-1;i>=0;i--){
		updateWeights(N->L[i]);
	}
}

void layerForward(const Matrix *M,const Array *in,Array *out,Array *deriv){
	int i,j;
	float a,tmp;
	for(j=0;j<M->n;j++){
		a=0;
		for(i=0;i<M->m;i++){
			a+=M->el[j*M->m+i]*in->el[i];
		}
		tmp=tanh(a);
		out->el[j]=tmp;
		deriv->el[j]=1.0-tmp*tmp;
	}
}

Array *nnForward(Net *N){
	int i;
	Matrix *M;
	Array *in;
	Array *out;
	Array *deriv;
	for(i=0;i<LAYERS;i++){
		M=N->L[i]->M;
		//in=N->L[i]->in;
		out=N->L[i]->out;
		deriv=N->L[i]->deriv;
//PRINTARRAY(in);
		layerForward(M,in,out,deriv);
//PRINTARRAY(out);
//		MatMul(*N->L[i]->M,*N->L[i]->in,*N->L[i]->out,*N->L[i]->deriv);
	}
	return out;
}

//this actually does more than simply multiply.
__global__ void MatMulKernel(const Matrix A, const Array x, Array y, Array deriv){
	int i;
	float Cval=0;
	//int row = blockIdx.y * blockDim.y + threadIdx.y;
  //int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
	for(i=0;i<A.m;i++){
		Cval+=A.el[row*A.m+i]*x.el[i];
	}
	float tmp=tanhf(Cval);
	y.el[row]=tmp;
	deriv.el[row]=1.0-(tmp*tmp);
}
/*
void MatMul(const Matrix A, const Array x, Array y, Array deriv)
{
    Matrix d_A;
    d_A.m = d_A.m = A.m; d_A.n = A.n;
    size_t size = A.m * A.n * sizeof(float);
    hipMalloc(&d_A.el, size);
    hipMemcpy(d_A.el, A.el, size,
               hipMemcpyHostToDevice);

		Array d_x;
		d_x.len=x.len;
		d_x.el=x.el;
		size_t xSz=x.len*sizeof(float);
    hipMalloc(&d_x.el,xSz);
    hipMemcpy(d_x.el,x.el,xSz,
	    hipMemcpyHostToDevice);
		
		Array d_y;
		d_y.len=y.len;
		d_y.el=y.el;
		size_t ySz=y.len*sizeof(float);
    hipMalloc(&d_y.el,ySz);
		
		Array d_deriv;
		d_deriv.len=deriv.len;
		d_deriv.el=deriv.el;
    hipMalloc(&d_deriv.el,ySz);

    // Invoke kernel
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(B.m / dimBlock.x, A.n / dimBlock.y);
    //dim3 dimGrid(dimBlock.x, A.n / dimBlock.y);
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(A.n / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, d_deriv);
    //MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y);

    // Read from device memory
    hipMemcpy(y.el, d_y.el, ySz,
    	hipMemcpyDeviceToHost);
    
		hipMemcpy(deriv.el, d_deriv.el, ySz,
    	hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.el);
    hipFree(d_x.el);
    hipFree(d_y.el);
    hipFree(d_deriv.el);
}
*/
// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.el[row * A.m + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.el[row * A.m + col] = value;
}
/*
// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.m    = BLOCK_SIZE;
    Asub.n   = BLOCK_SIZE;
    Asub.m   = A.m;
    Asub.el = &A.el[A.m * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.m = d_A.m = A.m; d_A.n = A.n;
    size_t size = A.m * A.n * sizeof(float);
    hipMalloc(&d_A.el, size);
    hipMemcpy(d_A.el, A.el, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.m = d_B.m = B.m; d_B.n = B.n;
    size = B.m * B.n * sizeof(float);

    hipMalloc(&d_B.el, size);
    hipMemcpy(d_B.el, B.el, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.m = d_C.m = C.m; d_C.n = C.n;
    size = C.m * C.n * sizeof(float);
    hipMalloc(&d_C.el, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.m / dimBlock.x, A.n / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.el, d_C.el, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.el);
    hipFree(d_B.el);
    hipFree(d_C.el);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.m / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}
*/
